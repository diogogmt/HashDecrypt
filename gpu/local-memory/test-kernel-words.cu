#include <iostream>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
// #include "md5.cuh"
// #include "cuPrintf.cu"





typedef unsigned char md5_byte_t; /* 8-bit byte */
typedef unsigned int md5_word_t; /* 32-bit word */

/* Define the state of the MD5 Algorithm. */
typedef struct md5_state_s {
    md5_word_t count[2];  /* message length in bits, lsw first */
    md5_word_t abcd[4];   /* digest buffer */
    md5_byte_t buf[64];   /* accumulate block */
} md5_state_t;









#undef BYTE_ORDER /* 1 = big-endian, -1 = little-endian, 0 = unknown */
#ifdef ARCH_IS_BIG_ENDIAN
	#define BYTE_ORDER (ARCH_IS_BIG_ENDIAN ? 1 : -1)
#else
	#define BYTE_ORDER 0
#endif

#define T_MASK ((md5_word_t)~0)
#define T1 /* 0xd76aa478 */ (T_MASK ^ 0x28955b87)
#define T2 /* 0xe8c7b756 */ (T_MASK ^ 0x173848a9)
#define T3    0x242070db
#define T4 /* 0xc1bdceee */ (T_MASK ^ 0x3e423111)
#define T5 /* 0xf57c0faf */ (T_MASK ^ 0x0a83f050)
#define T6    0x4787c62a
#define T7 /* 0xa8304613 */ (T_MASK ^ 0x57cfb9ec)
#define T8 /* 0xfd469501 */ (T_MASK ^ 0x02b96afe)
#define T9    0x698098d8
#define T10 /* 0x8b44f7af */ (T_MASK ^ 0x74bb0850)
#define T11 /* 0xffff5bb1 */ (T_MASK ^ 0x0000a44e)
#define T12 /* 0x895cd7be */ (T_MASK ^ 0x76a32841)
#define T13    0x6b901122
#define T14 /* 0xfd987193 */ (T_MASK ^ 0x02678e6c)
#define T15 /* 0xa679438e */ (T_MASK ^ 0x5986bc71)
#define T16    0x49b40821
#define T17 /* 0xf61e2562 */ (T_MASK ^ 0x09e1da9d)
#define T18 /* 0xc040b340 */ (T_MASK ^ 0x3fbf4cbf)
#define T19    0x265e5a51
#define T20 /* 0xe9b6c7aa */ (T_MASK ^ 0x16493855)
#define T21 /* 0xd62f105d */ (T_MASK ^ 0x29d0efa2)
#define T22    0x02441453
#define T23 /* 0xd8a1e681 */ (T_MASK ^ 0x275e197e)
#define T24 /* 0xe7d3fbc8 */ (T_MASK ^ 0x182c0437)
#define T25    0x21e1cde6
#define T26 /* 0xc33707d6 */ (T_MASK ^ 0x3cc8f829)
#define T27 /* 0xf4d50d87 */ (T_MASK ^ 0x0b2af278)
#define T28    0x455a14ed
#define T29 /* 0xa9e3e905 */ (T_MASK ^ 0x561c16fa)
#define T30 /* 0xfcefa3f8 */ (T_MASK ^ 0x03105c07)
#define T31    0x676f02d9
#define T32 /* 0x8d2a4c8a */ (T_MASK ^ 0x72d5b375)
#define T33 /* 0xfffa3942 */ (T_MASK ^ 0x0005c6bd)
#define T34 /* 0x8771f681 */ (T_MASK ^ 0x788e097e)
#define T35    0x6d9d6122
#define T36 /* 0xfde5380c */ (T_MASK ^ 0x021ac7f3)
#define T37 /* 0xa4beea44 */ (T_MASK ^ 0x5b4115bb)
#define T38    0x4bdecfa9
#define T39 /* 0xf6bb4b60 */ (T_MASK ^ 0x0944b49f)
#define T40 /* 0xbebfbc70 */ (T_MASK ^ 0x4140438f)
#define T41    0x289b7ec6
#define T42 /* 0xeaa127fa */ (T_MASK ^ 0x155ed805)
#define T43 /* 0xd4ef3085 */ (T_MASK ^ 0x2b10cf7a)
#define T44    0x04881d05
#define T45 /* 0xd9d4d039 */ (T_MASK ^ 0x262b2fc6)
#define T46 /* 0xe6db99e5 */ (T_MASK ^ 0x1924661a)
#define T47    0x1fa27cf8
#define T48 /* 0xc4ac5665 */ (T_MASK ^ 0x3b53a99a)
#define T49 /* 0xf4292244 */ (T_MASK ^ 0x0bd6ddbb)
#define T50    0x432aff97
#define T51 /* 0xab9423a7 */ (T_MASK ^ 0x546bdc58)
#define T52 /* 0xfc93a039 */ (T_MASK ^ 0x036c5fc6)
#define T53    0x655b59c3
#define T54 /* 0x8f0ccc92 */ (T_MASK ^ 0x70f3336d)
#define T55 /* 0xffeff47d */ (T_MASK ^ 0x00100b82)
#define T56 /* 0x85845dd1 */ (T_MASK ^ 0x7a7ba22e)
#define T57    0x6fa87e4f
#define T58 /* 0xfe2ce6e0 */ (T_MASK ^ 0x01d3191f)
#define T59 /* 0xa3014314 */ (T_MASK ^ 0x5cfebceb)
#define T60    0x4e0811a1
#define T61 /* 0xf7537e82 */ (T_MASK ^ 0x08ac817d)
#define T62 /* 0xbd3af235 */ (T_MASK ^ 0x42c50dca)
#define T63    0x2ad7d2bb
#define T64 /* 0xeb86d391 */ (T_MASK ^ 0x14792c6e)


__host__ __device__
static void md5_process(md5_state_t *pms, const md5_byte_t *data /*[64]*/) {
  md5_word_t a = pms->abcd[0], b = pms->abcd[1], c = pms->abcd[2], d = pms->abcd[3];
  md5_word_t t;

  /* Define storage for little-endian or both types of CPUs. */
  md5_word_t xbuf[16];
  const md5_word_t *X;

  {
    /*
     * Determine dynamically whether this is a big-endian or
     * little-endian machine, since we can use a more efficient
     * algorithm on the latter.
     */
    const int w = 1;

    if (*((const md5_byte_t *)&w)) /* dynamic little-endian */ {
      /*
      * On little-endian machines, we can process properly aligned
      * data without copying it.
      */
      if (!((data - (const md5_byte_t *)0) & 3)) {
        /* data are properly aligned */
        X = (const md5_word_t *)data;
      } else {
        /* not aligned */
        memcpy(xbuf, data, 64);
        X = xbuf;
      }
    } 
  }

  #define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32 - (n))))

  /* Round 1. */
  /* Let [abcd k s i] denote the operation
     a = b + ((a + F(b,c,d) + X[k] + T[i]) <<< s). */
  #define F(x, y, z) (((x) & (y)) | (~(x) & (z)))
  #define SET(a, b, c, d, k, s, Ti)\
    t = a + F(b,c,d) + X[k] + Ti;\
    a = ROTATE_LEFT(t, s) + b
    /* Do the following 16 operations. */
    SET(a, b, c, d,  0,  7,  T1);
    SET(d, a, b, c,  1, 12,  T2);
    SET(c, d, a, b,  2, 17,  T3);
    SET(b, c, d, a,  3, 22,  T4);
    SET(a, b, c, d,  4,  7,  T5);
    SET(d, a, b, c,  5, 12,  T6);
    SET(c, d, a, b,  6, 17,  T7);
    SET(b, c, d, a,  7, 22,  T8);
    SET(a, b, c, d,  8,  7,  T9);
    SET(d, a, b, c,  9, 12, T10);
    SET(c, d, a, b, 10, 17, T11);
    SET(b, c, d, a, 11, 22, T12);
    SET(a, b, c, d, 12,  7, T13);
    SET(d, a, b, c, 13, 12, T14);
    SET(c, d, a, b, 14, 17, T15);
    SET(b, c, d, a, 15, 22, T16);
  #undef SET

  /* Round 2. */
  /* Let [abcd k s i] denote the operation
     a = b + ((a + G(b,c,d) + X[k] + T[i]) <<< s). */
  #define G(x, y, z) (((x) & (z)) | ((y) & ~(z)))
  #define SET(a, b, c, d, k, s, Ti)\
    t = a + G(b,c,d) + X[k] + Ti;\
    a = ROTATE_LEFT(t, s) + b
   /* Do the following 16 operations. */
  SET(a, b, c, d,  1,  5, T17);
  SET(d, a, b, c,  6,  9, T18);
  SET(c, d, a, b, 11, 14, T19);
  SET(b, c, d, a,  0, 20, T20);
  SET(a, b, c, d,  5,  5, T21);
  SET(d, a, b, c, 10,  9, T22);
  SET(c, d, a, b, 15, 14, T23);
  SET(b, c, d, a,  4, 20, T24);
  SET(a, b, c, d,  9,  5, T25);
  SET(d, a, b, c, 14,  9, T26);
  SET(c, d, a, b,  3, 14, T27);
  SET(b, c, d, a,  8, 20, T28);
  SET(a, b, c, d, 13,  5, T29);
  SET(d, a, b, c,  2,  9, T30);
  SET(c, d, a, b,  7, 14, T31);
  SET(b, c, d, a, 12, 20, T32);
  #undef SET

  /* Round 3. */
  /* Let [abcd k s t] denote the operation
     a = b + ((a + H(b,c,d) + X[k] + T[i]) <<< s). */
  #define H(x, y, z) ((x) ^ (y) ^ (z))
  #define SET(a, b, c, d, k, s, Ti)\
    t = a + H(b,c,d) + X[k] + Ti;\
    a = ROTATE_LEFT(t, s) + b
  /* Do the following 16 operations. */
  SET(a, b, c, d,  5,  4, T33);
  SET(d, a, b, c,  8, 11, T34);
  SET(c, d, a, b, 11, 16, T35);
  SET(b, c, d, a, 14, 23, T36);
  SET(a, b, c, d,  1,  4, T37);
  SET(d, a, b, c,  4, 11, T38);
  SET(c, d, a, b,  7, 16, T39);
  SET(b, c, d, a, 10, 23, T40);
  SET(a, b, c, d, 13,  4, T41);
  SET(d, a, b, c,  0, 11, T42);
  SET(c, d, a, b,  3, 16, T43);
  SET(b, c, d, a,  6, 23, T44);
  SET(a, b, c, d,  9,  4, T45);
  SET(d, a, b, c, 12, 11, T46);
  SET(c, d, a, b, 15, 16, T47);
  SET(b, c, d, a,  2, 23, T48);
  #undef SET

  /* Round 4. */
  /* Let [abcd k s t] denote the operation
     a = b + ((a + I(b,c,d) + X[k] + T[i]) <<< s). */
  #define I(x, y, z) ((y) ^ ((x) | ~(z)))
  #define SET(a, b, c, d, k, s, Ti)\
    t = a + I(b,c,d) + X[k] + Ti;\
    a = ROTATE_LEFT(t, s) + b
  /* Do the following 16 operations. */
  SET(a, b, c, d,  0,  6, T49);
  SET(d, a, b, c,  7, 10, T50);
  SET(c, d, a, b, 14, 15, T51);
  SET(b, c, d, a,  5, 21, T52);
  SET(a, b, c, d, 12,  6, T53);
  SET(d, a, b, c,  3, 10, T54);
  SET(c, d, a, b, 10, 15, T55);
  SET(b, c, d, a,  1, 21, T56);
  SET(a, b, c, d,  8,  6, T57);
  SET(d, a, b, c, 15, 10, T58);
  SET(c, d, a, b,  6, 15, T59);
  SET(b, c, d, a, 13, 21, T60);
  SET(a, b, c, d,  4,  6, T61);
  SET(d, a, b, c, 11, 10, T62);
  SET(c, d, a, b,  2, 15, T63);
  SET(b, c, d, a,  9, 21, T64);
  #undef SET

  /* Then perform the following additions. (That is increment each
     of the four registers by the value it had before this block
     was started.) */
  pms->abcd[0] += a;
  pms->abcd[1] += b;
  pms->abcd[2] += c;
  pms->abcd[3] += d;
}


__host__ __device__
void md5_init(md5_state_t *pms) {
  pms->count[0] = pms->count[1] = 0;
  pms->abcd[0] = 0x67452301;
  pms->abcd[1] = /*0xefcdab89*/ T_MASK ^ 0x10325476;
  pms->abcd[2] = /*0x98badcfe*/ T_MASK ^ 0x67452301;
  pms->abcd[3] = 0x10325476;
}

__host__ __device__
void
md5_append(md5_state_t *pms, const md5_byte_t *data, int nbytes) {
  const md5_byte_t *p = data;
  int left = nbytes;
  int offset = (pms->count[0] >> 3) & 63;
  md5_word_t nbits = (md5_word_t)(nbytes << 3);

  if (nbytes <= 0) {
    return;
  }

  /* Update the message length. */
  pms->count[1] += nbytes >> 29;
  pms->count[0] += nbits;
  if (pms->count[0] < nbits) {
    pms->count[1]++;
  }

  /* Process an initial partial block. */
  if (offset) {
    int copy = (offset + nbytes > 64 ? 64 - offset : nbytes);

    memcpy(pms->buf + offset, p, copy);
    if (offset + copy < 64) {
      return;
    }
    p += copy;
    left -= copy;
    md5_process(pms, pms->buf);
  }

  /* Process full blocks. */
  for (; left >= 64; p += 64, left -= 64) {
    md5_process(pms, p);
  }

  /* Process a final partial block. */
  if (left) {
    memcpy(pms->buf, p, left);
  }
}

__host__ __device__ void
md5_finish(md5_state_t *pms, md5_byte_t digest[16]) {
  // static const md5_byte_t pad[64] = {
  const md5_byte_t pad[64] = {
    0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
  };
  md5_byte_t data[8];
  int i;

  /* Save the length before padding. */
  for (i = 0; i < 8; ++i) {
    data[i] = (md5_byte_t)(pms->count[i >> 2] >> ((i & 3) << 3));
  }
  /* Pad to 56 bytes mod 64. */
  md5_append(pms, pad, ((55 - (pms->count[0] >> 3)) & 63) + 1);
  /* Append the length. */
  md5_append(pms, data, 8);
  for (i = 0; i < 16; ++i) {
    digest[i] = (md5_byte_t)(pms->abcd[i >> 2] >> ((i & 3) << 3));
  }
}











int hex_to_decimal (char c) {
  switch (c) {
    case '0':
      return 0;
    case '1':
      return 1;
    case '2':
      return 2;
    case '3':
      return 3;
    case '4':
      return 4;
    case '5':
      return 5;
    case '6':
      return 6;
    case '7':
      return 7;
    case '8':
      return 8;
    case '9':
      return 9;
    case 'a':
    case 'A':
      return 10;
    case 'b':
    case 'B':
      return 11;
    case 'c':
    case 'C':
      return 12;
    case 'd':
    case 'D':
      return 13;
    case 'e':
    case 'E':
      return 14;
    case 'f':
    case 'F':
      return 15;
    default:
      fprintf(stdout, "FAILED to get convert %c to decimal\n", c);
      break;
  }

  return -1;
}

void break_down_hash (char* hash_str, md5_byte_t* hash) {
  int i, j;
  int digest = 0;
  int dec1 = 0;
  int dec2 = 0;
  for (i = 0, j = 0; i < 32; i += 2, j++) {
    dec1 = hex_to_decimal(hash_str[i]);
    dec2 = hex_to_decimal(hash_str[i+1]);
    // fprintf(stdout, "dec1: %d\n", dec1);
    // fprintf(stdout, "dec2: %d\n", dec2);
    digest =  dec1 * 16 + dec2;
    hash[j] = digest;
    // fprintf(stdout, "i %d - digest: %d\n", i, digest);
  }
}


// Create hash string for |word|
// This is the hash string to the original hash we are trying to break
// We use the hash string to create the 16 8bit hexadecinal chunks
void create_md5_hash_str(const char* word, char* hash_str) {
  // char hex_output[16*2 + 1];
  int di;

  md5_state_t state;
  md5_byte_t digest[16];  

  md5_init(&state);
  md5_append(&state, (const md5_byte_t *)word, strlen(word));
  md5_finish(&state, digest);
  
  for (di = 0; di < 16; di++) {
    sprintf(hash_str + di * 2, "%02x", digest[di]); 
  }
}



__global__ void testKernel (md5_byte_t* hash_to_break, char* hash_word) {

	char di = 0;
  md5_byte_t cached_hash[16];
  for (di = 0; di < 16; di++) {
    cached_hash[di] = hash_to_break[di];
  }


  //  __shared__ const md5_byte_t pad[64] = {
  //    0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
  //    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
  //    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
  //    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
  //  };
    __shared__ md5_byte_t pad[64];
    pad[0] = 0x80;
    for (di = 1; di < 64; di++) {
      pad[di] = 0;
    }

  char word[4];
  word[0] = (char) threadIdx.x + 32;
  word[1] = (char) 32;
  word[2] = (char) 32;
  word[3] = (char) 32;

  // We need to print here because cuPrintf doesnt work
  // if any cuPrintfs are inside a control structure statements such as if statement os for loops
  // cuPrintf("%c\n", word[0]);

  char i_1, i_2, i_3;

  char cached_hash_word[4];
  cached_hash_word[0] = (char)0;

  md5_state_t state;
  md5_byte_t digest[16]; 
  char counter = 0;

  for (i_1 = 32; i_1 < 42; i_1++) {
    word[1] = (char) i_1;
    for (i_2 = 32; i_2 < 42; i_2++) {
      word[2] = (char) i_2;
      for (i_3 = 32; i_3 < 42; i_3++) {
        word[3] = (char) i_3;
        //md5_init(&state);
        state.count[0] = state.count[1] = 0;
        state.abcd[0] = 0x67452301;
        state.abcd[1] = /*0xefcdab89*/ T_MASK ^ 0x10325476;
        state.abcd[2] = /*0x98badcfe*/ T_MASK ^ 0x67452301;
        state.abcd[3] = 0x10325476;
        // end md5_init

        md5_append(&state, (const md5_byte_t *)word, 4);
        //md5_finish(&state, digest);
        //md5_finish(md5_state_t *pms, md5_byte_t digest[16]) {
        // static const md5_byte_t pad[64] = {
        // const md5_byte_t pad[64] = {
        //   0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        //   0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        //   0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        //   0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
        // };
        md5_byte_t data[8];
        int i;

        /* Save the length before padding. */
        for (i = 0; i < 8; ++i) {
          data[i] = (md5_byte_t)(state.count[i >> 2] >> ((i & 3) << 3));
        }
        /* Pad to 56 bytes mod 64. */
        md5_append(&state, pad, ((55 - (state.count[0] >> 3)) & 63) + 1);
        /* Append the length. */
        md5_append(&state, data, 8);
        for (i = 0; i < 16; ++i) {
          digest[i] = (md5_byte_t)(state.abcd[i >> 2] >> ((i & 3) << 3));
        }
        // end md5_finish

        counter = 0;
        for (di = 0; di < 16; di++) {
          if (digest[di] == cached_hash[di]) {
            counter++;
          }
        }
        if (counter == 16) {
        	for (di = 0; di < 4; di++) {
        		cached_hash_word[di] = word[di];
        	}
        }
      } // END Loop 3
    } // END Loop 2
  } // END Loop 1
  if (cached_hash_word[0] != (char)0) {
	  for (di = 0; di < 4; di++) {
			hash_word[di] = cached_hash_word[di];
		}
  }
} // END kernel

int main (int argc, char *argv[]) {
  if (argc != 2) {
    printf("**invalid number of arguments**\n");
    return 1;
  }

  char hash_str[32];
  md5_byte_t h_hash[16];

  create_md5_hash_str(argv[1], hash_str);

  printf("original_word: %s\n", argv[1]);
  printf("hash to break: %s\n", hash_str);

  break_down_hash(hash_str, h_hash);

  // Declare device hash
  md5_byte_t* d_hash;
  hipMalloc((void**)&d_hash, sizeof(md5_byte_t) * 16);
  hipMemcpy(d_hash, h_hash, sizeof(md5_byte_t) * 16, hipMemcpyHostToDevice);

  // Declare device hash
char* d_word;
hipMalloc((void**)&d_word, sizeof(char) * 4);

  // cudaPrintfInit();

  testKernel<<<1,10>>>(d_hash, d_word);

  // cudaPrintfDisplay(stdout, true);
  // cudaPrintfEnd();
  char h_word[4];
  hipMemcpy(h_word, d_word, sizeof(char) * 4, hipMemcpyDeviceToHost);
  printf("|%c%c%c%c|\n", h_word[0], h_word[1], h_word[2], h_word[3]);

  hipFree(d_hash);
  hipFree(d_word);

  printf("finished!\n");
}



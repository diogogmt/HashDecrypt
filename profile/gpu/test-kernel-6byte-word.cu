#include "hip/hip_runtime.h"
#include "../libs/md5.h"
#include <iostream>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#include "../libs/cuPrintf.cu"



void create_md5_hash_str (const char *word, char *hash_str);
void break_down_hash (md5_byte_t *hash, char *hash_str);
int hex_to_decimal (char c);


#define T_MASK ((md5_word_t)~0)
#define T1 /* 0xd76aa478 */ (T_MASK ^ 0x28955b87)
#define T2 /* 0xe8c7b756 */ (T_MASK ^ 0x173848a9)
#define T3    0x242070db
#define T4 /* 0xc1bdceee */ (T_MASK ^ 0x3e423111)
#define T5 /* 0xf57c0faf */ (T_MASK ^ 0x0a83f050)
#define T6    0x4787c62a
#define T7 /* 0xa8304613 */ (T_MASK ^ 0x57cfb9ec)
#define T8 /* 0xfd469501 */ (T_MASK ^ 0x02b96afe)
#define T9    0x698098d8
#define T10 /* 0x8b44f7af */ (T_MASK ^ 0x74bb0850)
#define T11 /* 0xffff5bb1 */ (T_MASK ^ 0x0000a44e)
#define T12 /* 0x895cd7be */ (T_MASK ^ 0x76a32841)
#define T13    0x6b901122
#define T14 /* 0xfd987193 */ (T_MASK ^ 0x02678e6c)
#define T15 /* 0xa679438e */ (T_MASK ^ 0x5986bc71)
#define T16    0x49b40821
#define T17 /* 0xf61e2562 */ (T_MASK ^ 0x09e1da9d)
#define T18 /* 0xc040b340 */ (T_MASK ^ 0x3fbf4cbf)
#define T19    0x265e5a51
#define T20 /* 0xe9b6c7aa */ (T_MASK ^ 0x16493855)
#define T21 /* 0xd62f105d */ (T_MASK ^ 0x29d0efa2)
#define T22    0x02441453
#define T23 /* 0xd8a1e681 */ (T_MASK ^ 0x275e197e)
#define T24 /* 0xe7d3fbc8 */ (T_MASK ^ 0x182c0437)
#define T25    0x21e1cde6
#define T26 /* 0xc33707d6 */ (T_MASK ^ 0x3cc8f829)
#define T27 /* 0xf4d50d87 */ (T_MASK ^ 0x0b2af278)
#define T28    0x455a14ed
#define T29 /* 0xa9e3e905 */ (T_MASK ^ 0x561c16fa)
#define T30 /* 0xfcefa3f8 */ (T_MASK ^ 0x03105c07)
#define T31    0x676f02d9
#define T32 /* 0x8d2a4c8a */ (T_MASK ^ 0x72d5b375)
#define T33 /* 0xfffa3942 */ (T_MASK ^ 0x0005c6bd)
#define T34 /* 0x8771f681 */ (T_MASK ^ 0x788e097e)
#define T35    0x6d9d6122
#define T36 /* 0xfde5380c */ (T_MASK ^ 0x021ac7f3)
#define T37 /* 0xa4beea44 */ (T_MASK ^ 0x5b4115bb)
#define T38    0x4bdecfa9
#define T39 /* 0xf6bb4b60 */ (T_MASK ^ 0x0944b49f)
#define T40 /* 0xbebfbc70 */ (T_MASK ^ 0x4140438f)
#define T41    0x289b7ec6
#define T42 /* 0xeaa127fa */ (T_MASK ^ 0x155ed805)
#define T43 /* 0xd4ef3085 */ (T_MASK ^ 0x2b10cf7a)
#define T44    0x04881d05
#define T45 /* 0xd9d4d039 */ (T_MASK ^ 0x262b2fc6)
#define T46 /* 0xe6db99e5 */ (T_MASK ^ 0x1924661a)
#define T47    0x1fa27cf8
#define T48 /* 0xc4ac5665 */ (T_MASK ^ 0x3b53a99a)
#define T49 /* 0xf4292244 */ (T_MASK ^ 0x0bd6ddbb)
#define T50    0x432aff97
#define T51 /* 0xab9423a7 */ (T_MASK ^ 0x546bdc58)
#define T52 /* 0xfc93a039 */ (T_MASK ^ 0x036c5fc6)
#define T53    0x655b59c3
#define T54 /* 0x8f0ccc92 */ (T_MASK ^ 0x70f3336d)
#define T55 /* 0xffeff47d */ (T_MASK ^ 0x00100b82)
#define T56 /* 0x85845dd1 */ (T_MASK ^ 0x7a7ba22e)
#define T57    0x6fa87e4f
#define T58 /* 0xfe2ce6e0 */ (T_MASK ^ 0x01d3191f)
#define T59 /* 0xa3014314 */ (T_MASK ^ 0x5cfebceb)
#define T60    0x4e0811a1
#define T61 /* 0xf7537e82 */ (T_MASK ^ 0x08ac817d)
#define T62 /* 0xbd3af235 */ (T_MASK ^ 0x42c50dca)
#define T63    0x2ad7d2bb
#define T64 /* 0xeb86d391 */ (T_MASK ^ 0x14792c6e)


#define STR_SIZE 6
// If the string has 4 chars, it has a size of 32 bits
// X is a unsigned int pointer, it points to 32 bit chuncks of data
// X[0] will be the whole 4 byte string
// the byte right after the string has the value 0x80, which was translating to 128 since the first nibble is 0 and the second is 8
// a byte with the second nibble as 8 is the decimal 128 --> 1000 0000
// the formula for X_14 is STR_SIZE << 3 as long the string has less than 32 bytes
// after that the formula is (STR_SIZE - 32) << 3 and X_15 will be 1
// We'll be focusing on string less than 32 bytes for now
#define X_1 128
// #define X_14 (STR_SIZE << 3)
#define X_14 48
#define ZERO 0
__global__ void do_md5(md5_word_t* hash_to_break, md5_byte_t* hash_word, md5_byte_t* ascii_codes,
  char start_loop, char end_loop) {

  char it;
  __shared__ md5_word_t cached_hash[16];
  for (it = 0; it < 4; it++) {
    cached_hash[it] = hash_to_break[it];
  }

  __shared__ md5_byte_t cached_ascii_codes[64];
  for (it = 0; it < 64; it++) {
    cached_ascii_codes[it] = ascii_codes[it];
  }

  md5_word_t a;
  md5_word_t b;
  md5_word_t c;
  md5_word_t d;

  md5_word_t t;

  char word[8];
  word[0] = cached_ascii_codes[threadIdx.x];
  word[6] = 0x80;
  word[7] = 0;
  
  char i_1, i_2, i_3, i_4, i_5;


  char begin_1 = 0 + (8 * threadIdx.y);
  char end_1 = 8 + (8 * threadIdx.y);

  char begin_2 = 0;
  char end_2 = 64;

  char cached_start_loop = start_loop;
  char cached_end_loop = end_loop;

  const md5_word_t *X; 

  // unsigned counter = 0;
  for (i_1 = begin_1; i_1 < end_1; i_1++) {
    word[1] = cached_ascii_codes[i_1];
    for (i_2 = begin_2; i_2 < end_2; i_2++) {
      word[2] = cached_ascii_codes[i_2];
      for (i_3 = 0; i_3 < 64; i_3++) {
        word[3] = cached_ascii_codes[i_3];
        for (i_4 = 0; i_4 < 4; i_4++) {
          word[4] = cached_ascii_codes[i_4];
          for (i_5 = cached_start_loop; i_5 < cached_end_loop; i_5++) {
            word[5] = cached_ascii_codes[i_5];

            // counter++;

            X = (const md5_word_t *)word;
            a = 0x67452301;
            b = /*0xefcdab89*/ T_MASK ^ 0x10325476;
            c = /*0x98badcfe*/ T_MASK ^ 0x67452301;
            d = 0x10325476;
            /* Round 1. */
            /* Let [abcd k s i] denote the operation
               a = b + ((a + F(b,c,d) + X[k] + T[i]) <<< s).
            */
            /* Do the following 16 operations. */
            // Set 1
            t = a + ((b & c) | (~b & d)) + X[0] + T1; a = ((t << 7) | (t >> (32 - 7))) + b;

            t = d + ((a & b) | (~a & c)) + X[1] + T2; d = ((t << 12) | (t >> (32 - 12))) + a;

            t = c + ((d & a) | (~d & b)) + T3; c = ((t << 17) | (t >> (32 - 17))) + d;

            t = b + ((c & d) | (~c & a)) + T4; b = ((t << 22) | (t >> (32 - 22))) + c;

            // Set 2
            t = a + ((b & c) | (~b & d)) + T5; a = ((t << 7) | (t >> (32 - 7))) + b;

            t = d + ((a & b) | (~a & c)) + T6; d = ((t << 12) | (t >> (32 - 12))) + a;

            t = c + ((d & a) | (~d & b)) + T7; c = ((t << 17) | (t >> (32 - 17))) + d;

            t = b + ((c & d) | (~c & a)) + T8; b = ((t << 22) | (t >> (32 - 22))) + c;

            // Set 3
            t = a + ((b & c) | (~b & d)) + T9; a = ((t << 7) | (t >> (32 - 7))) + b;

            t = d + ((a & b) | (~a & c)) + T10; d = ((t << 12) | (t >> (32 - 12))) + a;

            t = c + ((d & a) | (~d & b)) + T11; c = ((t << 17) | (t >> (32 - 17))) + d;

            t = b + ((c & d) | (~c & a)) + T12; b = ((t << 22) | (t >> (32 - 22))) + c;

            // Set 4
            t = a + ((b & c) | (~b & d)) + T13; a = ((t << 7) | (t >> (32 - 7))) + b;

            t = d + ((a & b) | (~a & c)) + T14; d = ((t << 12) | (t >> (32 - 12))) + a;

            t = c + ((d & a) | (~d & b)) + X_14 + T15; c = ((t << 17) | (t >> (32 - 17))) + d;

            t = b + ((c & d) | (~c & a)) + T16; b = ((t << 22) | (t >> (32 - 22))) + c;



            /* Round 2. */
            /* Let [abcd k s i] denote the operation
               a = b + ((a + G(b,c,d) + X[k] + T[i]) <<< s).
            */
            /* Do the following 16 operations. */
            // Set 1
            t = a + ((b & d) | (c & ~d)) + X[1] + T17; a = ((t << 5) | (t >> (32 - 5))) + b;

            t = d + ((a & c) | (b & ~c)) + T18; d = ((t << 9) | (t >> (32 - 9))) + a;

            t = c + ((d & b) | (a & ~b)) + T19; c = ((t << 14) | (t >> (32 - 14))) + d;

            t = b + ((c & a) | (d & ~a)) + X[0] + T20; b = ((t << 20) | (t >> (32 - 20))) + c;

            // Set 2
            t = a + ((b & d) | (c & ~d)) + T21; a = ((t << 5) | (t >> (32 - 5))) + b;

            t = d + ((a & c) | (b & ~c)) + T22; d = ((t << 9) | (t >> (32 - 9))) + a;

            t = c + ((d & b) | (a & ~b)) + T23; c = ((t << 14) | (t >> (32 - 14))) + d;

            t = b + ((c & a) | (d & ~a)) + T24; b = ((t << 20) | (t >> (32 - 20))) + c;

            // Set 3
            t = a + ((b & d) | (c & ~d)) + T25; a = ((t << 5) | (t >> (32 - 5))) + b;

            t = d + ((a & c) | (b & ~c)) + X_14 + T26; d = ((t << 9) | (t >> (32 - 9))) + a;

            t = c + ((d & b) | (a & ~b)) + T27; c = ((t << 14) | (t >> (32 - 14))) + d;

            t = b + ((c & a) | (d & ~a)) + T28; b = ((t << 20) | (t >> (32 - 20))) + c;

            // Set 4
            t = a + ((b & d) | (c & ~d)) + T29; a = ((t << 5) | (t >> (32 - 5))) + b;

            t = d + ((a & c) | (b & ~c)) + T30; d = ((t << 9) | (t >> (32 - 9))) + a;

            t = c + ((d & b) | (a & ~b)) + T31; c = ((t << 14) | (t >> (32 - 14))) + d;

            t = b + ((c & a) | (d & ~a)) + T32; b = ((t << 20) | (t >> (32 - 20))) + c;




            /* Round 3. */
            /* Let [abcd k s t] denote the operation
               a = b + ((a + H(b,c,d) + X[k] + T[i]) <<< s).
            */
            /* Do the following 16 operations. */
            // Set 1
            t = a + (b ^ c ^ d) + T33; a = ((t << 4) | (t >> (32 - 4))) + b;

            t = d + (a ^ b ^ c) + T34; d = ((t << 11) | (t >> (32 - 11))) + a;

            t = c + (d ^ a ^ b) + T35; c = ((t << 16) | (t >> (32 - 16))) + d;

            t = b + (c ^ d ^ a) + X_14 + T36; b = ((t << 23) | (t >> (32 - 23))) + c;

            // Set 2
            t = a + (b ^ c ^ d) + X[1] + T37; a = ((t << 4) | (t >> (32 - 4))) + b;

            t = d + (a ^ b ^ c) + T38; d = ((t << 11) | (t >> (32 - 11))) + a;

            t = c + (d ^ a ^ b) + T39; c = ((t << 16) | (t >> (32 - 16))) + d;

            t = b + (c ^ d ^ a) + T40; b = ((t << 23) | (t >> (32 - 23))) + c;

            // Set 3
            t = a + (b ^ c ^ d) + T41; a = ((t << 4) | (t >> (32 - 4))) + b;

            t = d + (a ^ b ^ c) + X[0] + T42; d = ((t << 11) | (t >> (32 - 11))) + a;

            t = c + (d ^ a ^ b) + T43; c = ((t << 16) | (t >> (32 - 16))) + d;

            t = b + (c ^ d ^ a) + T44; b = ((t << 23) | (t >> (32 - 23))) + c;

            // Set 4
            t = a + (b ^ c ^ d) + T45; a = ((t << 4) | (t >> (32 - 4))) + b;

            t = d + (a ^ b ^ c) + T46; d = ((t << 11) | (t >> (32 - 11))) + a;

            t = c + (d ^ a ^ b) + T47; c = ((t << 16) | (t >> (32 - 16))) + d;

            t = b + (c ^ d ^ a) + T48; b = ((t << 23) | (t >> (32 - 23))) + c;




            /* Round 4. */
            /* Let [abcd k s t] denote the operation
               a = b + ((a + I(b,c,d) + X[k] + T[i]) <<< s).
            */
            /* Do the following 16 operations. */
            // Set 1
            t = a + (c ^ (b | ~d)) + X[0] + T49; a = ((t << 6) | (t >> (32 - 6))) + b;

            t = d + (b ^ (a | ~c)) + T50; d = ((t << 10) | (t >> (32 - 10))) + a;

            t = c + (a ^ (d | ~b)) + X_14 + T51; c = ((t << 15) | (t >> (32 - 15))) + d;

            t = b + (d ^ (c | ~a)) + T52; b = ((t << 21) | (t >> (32 - 21))) + c;

            // Set 2
            t = a + (c ^ (b | ~d)) + T53; a = ((t << 6) | (t >> (32 - 6))) + b;

            t = d + (b ^ (a | ~c)) + T54; d = ((t << 10) | (t >> (32 - 10))) + a;

            t = c + (a ^ (d | ~b)) + T55; c = ((t << 15) | (t >> (32 - 15))) + d;

            t = b + (d ^ (c | ~a)) + X[1] + T56; b = ((t << 21) | (t >> (32 - 21))) + c;

            // Set 3
            t = a + (c ^ (b | ~d)) + T57; a = ((t << 6) | (t >> (32 - 6))) + b;

            t = d + (b ^ (a | ~c)) + T58; d = ((t << 10) | (t >> (32 - 10))) + a;

            t = c + (a ^ (d | ~b)) + T59; c = ((t << 15) | (t >> (32 - 15))) + d;

            t = b + (d ^ (c | ~a)) + T60; b = ((t << 21) | (t >> (32 - 21))) + c;

            // Set 4
            t = a + (c ^ (b | ~d)) + T61; a = ((t << 6) | (t >> (32 - 6))) + b;

            t = d + (b ^ (a | ~c)) + T62; d = ((t << 10) | (t >> (32 - 10))) + a;

            t = c + (a ^ (d | ~b)) + T63; c = ((t << 15) | (t >> (32 - 15))) + d;

            t = b + (d ^ (c | ~a)) + T64; b = ((t << 21) | (t >> (32 - 21))) + c;


            /* Then perform the following additions. (That is increment each
               of the four registers by the value it had before this block
               was started.)
            */
            a += 0x67452301;
            b += T_MASK ^ 0x10325476;
            c += T_MASK ^ 0x67452301;
            d += 0x10325476;
            
              if (cached_hash[0]  == a   &&
                  cached_hash[1]  == b   &&
                  cached_hash[2]  == c   &&
                  cached_hash[3]  == d
              ) {
              cuPrintf("\n**************************found*****************\n\n");              

              hash_word[0] = word[0];
              hash_word[1] = word[1];
              hash_word[2] = word[2];
              hash_word[3] = word[3];
              hash_word[4] = word[4];
              hash_word[5] = word[5];
              // cuPrintf("device word |%c%c%c%c%c%c|\n",
              //   hash_word[0], hash_word[1], hash_word[2], hash_word[3], hash_word[4], hash_word[5]);

            }
          } // END Loop 5
        } // END Loop 4
      } // END Loop 3
    } // END Loop 2
  } // END Loop 2
  // cuPrintf("counter: %lu - word[0]: %d - range1: %d- %d - range2: %d-%d\n",
  //   counter, word[0], begin_1, end_1, begin_2, end_2);
}





int main (int argc, char *argv[]) {
  fprintf(stdout, "main\n");

  if (argc != 2) {
    printf("**invalid number of arguments**\n");
    return 1;
  }

  char hash_str[32];
  md5_byte_t h_hash[16];

  // Generate a MD5 hash string for the word passed in as an arg
  // The generated hash is the one we'll try to break
  create_md5_hash_str(argv[1], hash_str);

  char word_to_break[7];

  strcpy(word_to_break, argv[1]);

  fprintf(stdout, "original_word: |%s|\n", word_to_break);
  fprintf(stdout, "hash to break: %s\n", hash_str);

  // Split the MD5 hash into 16 8bit char chunks
  // The idea is split the 32 bit char string into 16 chunks of 2 chars each
  // Then convert the ASCII value of the pair of chars to their hexadecimal
  // The reason being because when the MD5 hash is generated instead of creting a string
  // the hash is outputed in 16 chunks of hexadecimal value
  // So to increase the speed during comparasion we convert the hash we are trying to break to 16 chunks of hexadecimal
  // values so when it comes the time to compare the generated hashes with the hash that we are tying to break
  // we only need to issue a memcmp insetad of converting the bytes to a string and doing a strcmp
  break_down_hash(h_hash, hash_str);


  md5_word_t* X = (md5_word_t *)h_hash;
  printf("X[0]: %u\n", X[0]);
  printf("X[1]: %u\n", X[1]);
  printf("X[2]: %u\n", X[2]);
  printf("X[3]: %u\n", X[3]);

  hipError_t error;

  // Declare device hash
  md5_word_t* d_hash;
  error = hipMalloc((void**)&d_hash, sizeof(md5_word_t) * 4);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  error = hipMemcpy(d_hash, X, sizeof(md5_word_t) * 4, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  // Declare device word
  md5_byte_t* d_word;
  error = hipMalloc((void**)&d_word, sizeof(md5_byte_t) * 6);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }


  int ascci_counter = 0;
  md5_byte_t h_ascii_codes[64];
  for (int i = 48; i <= 57; i++) {
    h_ascii_codes[ascci_counter++] = i;
  }
  for (int i = 65; i <= 90; i++) {
    h_ascii_codes[ascci_counter++] = i;
  }
  for (int i = 97; i <= 122; i++) {
    h_ascii_codes[ascci_counter++] = i;
  }
  h_ascii_codes[62] = 63; // ?
  h_ascii_codes[63] = 64; // @

  for (int i = 0; i < 64; i++) {
    // printf("h_ascii_codes[%d]: %d\n", i, h_ascii_codes[i]);
  }

  // Declare device hash
  md5_byte_t* d_ascii_codes;
  error = hipMalloc((void**)&d_ascii_codes, sizeof(md5_byte_t) * 64);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  error = hipMemcpy(d_ascii_codes, h_ascii_codes, sizeof(md5_byte_t) * 64, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }  


  cudaPrintfInit();

  // dim3 dimBlock(64, 16, 1);
  // dim3 dimGrid(32, 1, 1);

  dim3 dimBlock(64, 8, 1);
  dim3 dimGrid(1, 1, 1);

  for (int i = 0; i < 64; i+=16) {
    printf("launching kernel...\n");
    do_md5<<<dimGrid, dimBlock>>>(d_hash, d_word, d_ascii_codes, 0+i, 8+i);
    printf("finished launching kernel...\n");

    // synchronize the device and the host
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
      printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
    }

    
    md5_byte_t h_word[6];
    error = hipMemcpy(h_word, d_word, sizeof(md5_byte_t) * 6, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
      printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
    }

    printf("device word |%c%c%c%c%c%c|\n", h_word[0], h_word[1], h_word[2], h_word[3], h_word[4], h_word[5]);
    printf("original    |%s|\n", word_to_break);

    if (!memcmp(word_to_break, h_word, sizeof(md5_byte_t) * 6)) {
      printf("Broke hash!\n");
      break;
    }
    printf("going to the next iteration...\n");
  }

  printf("finished!\n");

  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();

  error = hipFree(d_hash);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  error = hipFree(d_word);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  return 1;
}




// Create hash string for |word|
// This is the hash string to the original hash we are trying to break
// We use the hash string to create the 16 8bit hexadecinal chunks
void create_md5_hash_str(const char* word, char* hash_str) {
  int di;

  md5_state_t state;
  md5_byte_t digest[16];  

  md5_init(&state);
  md5_append(&state, (const md5_byte_t *)word, strlen(word));
  md5_finish(&state, digest);
  
  for (di = 0; di < 16; di++) {
    sprintf(hash_str + di * 2, "%02x", digest[di]); 
  }
}

void break_down_hash (md5_byte_t* hash, char* hash_str) {
  int i, j;
  int digest = 0;
  int dec1 = 0;
  int dec2 = 0;
  for (i = 0, j = 0; i < 32; i += 2, j++) {
    dec1 = hex_to_decimal(hash_str[i]);
    dec2 = hex_to_decimal(hash_str[i+1]);
    // fprintf(stdout, "dec1: %d\n", dec1);
    // fprintf(stdout, "dec2: %d\n", dec2);
    digest =  dec1 * 16 + dec2;
    hash[j] = digest;
    // fprintf(stdout, "i %d - digest: %d\n", i, digest);
  }
}

int hex_to_decimal (char c) {
  switch (c) {
    case '0':
      return 0;
    case '1':
      return 1;
    case '2':
      return 2;
    case '3':
      return 3;
    case '4':
      return 4;
    case '5':
      return 5;
    case '6':
      return 6;
    case '7':
      return 7;
    case '8':
      return 8;
    case '9':
      return 9;
    case 'a':
    case 'A':
      return 10;
    case 'b':
    case 'B':
      return 11;
    case 'c':
    case 'C':
      return 12;
    case 'd':
    case 'D':
      return 13;
    case 'e':
    case 'E':
      return 14;
    case 'f':
    case 'F':
      return 15;
    default:
      fprintf(stdout, "FAILED to get convert %c to decimal\n", c);
  }

  return -1;
}


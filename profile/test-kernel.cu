#include "hip/hip_runtime.h"
#include "md5.h"
#include <iostream>
#include <cmath>
#include <cstring>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#include "cuPrintf.cu"



void create_md5_hash_str (const char *word, char *hash_str);
void break_down_hash (md5_byte_t *hash, char *hash_str);
int hex_to_decimal (char c);


#define T_MASK ((md5_word_t)~0)
#define T1 /* 0xd76aa478 */ (T_MASK ^ 0x28955b87)
#define T2 /* 0xe8c7b756 */ (T_MASK ^ 0x173848a9)
#define T3    0x242070db
#define T4 /* 0xc1bdceee */ (T_MASK ^ 0x3e423111)
#define T5 /* 0xf57c0faf */ (T_MASK ^ 0x0a83f050)
#define T6    0x4787c62a
#define T7 /* 0xa8304613 */ (T_MASK ^ 0x57cfb9ec)
#define T8 /* 0xfd469501 */ (T_MASK ^ 0x02b96afe)
#define T9    0x698098d8
#define T10 /* 0x8b44f7af */ (T_MASK ^ 0x74bb0850)
#define T11 /* 0xffff5bb1 */ (T_MASK ^ 0x0000a44e)
#define T12 /* 0x895cd7be */ (T_MASK ^ 0x76a32841)
#define T13    0x6b901122
#define T14 /* 0xfd987193 */ (T_MASK ^ 0x02678e6c)
#define T15 /* 0xa679438e */ (T_MASK ^ 0x5986bc71)
#define T16    0x49b40821
#define T17 /* 0xf61e2562 */ (T_MASK ^ 0x09e1da9d)
#define T18 /* 0xc040b340 */ (T_MASK ^ 0x3fbf4cbf)
#define T19    0x265e5a51
#define T20 /* 0xe9b6c7aa */ (T_MASK ^ 0x16493855)
#define T21 /* 0xd62f105d */ (T_MASK ^ 0x29d0efa2)
#define T22    0x02441453
#define T23 /* 0xd8a1e681 */ (T_MASK ^ 0x275e197e)
#define T24 /* 0xe7d3fbc8 */ (T_MASK ^ 0x182c0437)
#define T25    0x21e1cde6
#define T26 /* 0xc33707d6 */ (T_MASK ^ 0x3cc8f829)
#define T27 /* 0xf4d50d87 */ (T_MASK ^ 0x0b2af278)
#define T28    0x455a14ed
#define T29 /* 0xa9e3e905 */ (T_MASK ^ 0x561c16fa)
#define T30 /* 0xfcefa3f8 */ (T_MASK ^ 0x03105c07)
#define T31    0x676f02d9
#define T32 /* 0x8d2a4c8a */ (T_MASK ^ 0x72d5b375)
#define T33 /* 0xfffa3942 */ (T_MASK ^ 0x0005c6bd)
#define T34 /* 0x8771f681 */ (T_MASK ^ 0x788e097e)
#define T35    0x6d9d6122
#define T36 /* 0xfde5380c */ (T_MASK ^ 0x021ac7f3)
#define T37 /* 0xa4beea44 */ (T_MASK ^ 0x5b4115bb)
#define T38    0x4bdecfa9
#define T39 /* 0xf6bb4b60 */ (T_MASK ^ 0x0944b49f)
#define T40 /* 0xbebfbc70 */ (T_MASK ^ 0x4140438f)
#define T41    0x289b7ec6
#define T42 /* 0xeaa127fa */ (T_MASK ^ 0x155ed805)
#define T43 /* 0xd4ef3085 */ (T_MASK ^ 0x2b10cf7a)
#define T44    0x04881d05
#define T45 /* 0xd9d4d039 */ (T_MASK ^ 0x262b2fc6)
#define T46 /* 0xe6db99e5 */ (T_MASK ^ 0x1924661a)
#define T47    0x1fa27cf8
#define T48 /* 0xc4ac5665 */ (T_MASK ^ 0x3b53a99a)
#define T49 /* 0xf4292244 */ (T_MASK ^ 0x0bd6ddbb)
#define T50    0x432aff97
#define T51 /* 0xab9423a7 */ (T_MASK ^ 0x546bdc58)
#define T52 /* 0xfc93a039 */ (T_MASK ^ 0x036c5fc6)
#define T53    0x655b59c3
#define T54 /* 0x8f0ccc92 */ (T_MASK ^ 0x70f3336d)
#define T55 /* 0xffeff47d */ (T_MASK ^ 0x00100b82)
#define T56 /* 0x85845dd1 */ (T_MASK ^ 0x7a7ba22e)
#define T57    0x6fa87e4f
#define T58 /* 0xfe2ce6e0 */ (T_MASK ^ 0x01d3191f)
#define T59 /* 0xa3014314 */ (T_MASK ^ 0x5cfebceb)
#define T60    0x4e0811a1
#define T61 /* 0xf7537e82 */ (T_MASK ^ 0x08ac817d)
#define T62 /* 0xbd3af235 */ (T_MASK ^ 0x42c50dca)
#define T63    0x2ad7d2bb
#define T64 /* 0xeb86d391 */ (T_MASK ^ 0x14792c6e)


#define STR_SIZE 4
// If the string has 4 chars, it has a size of 32 bits
// X is a unsigned int pointer, it points to 32 bit chuncks of data
// X[0] will be the whole 4 byte string
// the byte right after the string has the value 0x80, which was translating to 128 since the first nibble is 0 and the second is 8
// a byte with the second nibble as 8 is the decimal 128 --> 1000 0000
// the formula for X_14 is STR_SIZE << 3 as long the string has less than 32 bytes
// after that the formula is (STR_SIZE - 32) << 3 and X_15 will be 1
// We'll be focusing on string less than 32 bytes for now
#define X_1 128
// #define X_14 (STR_SIZE << 3)
#define X_14 32
#define ZERO 0
__global__ void do_md5(md5_byte_t* hash_to_break, md5_byte_t* hash_word) {

  __shared__ md5_byte_t cached_hash[16];
  for (char it = 0; it < 16; it++) {
    cached_hash[it] = hash_to_break[it];
  }

  md5_word_t a;
  md5_word_t b;
  md5_word_t c;
  md5_word_t d;

  md5_word_t t;

  md5_word_t word;

  cuPrintf("word: %u\n", threadIdx.x | 0x00F);

  char i_1, i_2, i_3;

  for (i_1 = 32; i_1 < 127; i_1++) {
    for (i_2 = 32; i_2 < 127; i_2++) {
      for (i_3 = 32; i_3 < 127; i_3++) {
        word = (threadIdx.x + 32) | (i_1 << 8) | (i_2 << 16) | (i_3 << 24);

        a = 0x67452301;
        b = /*0xefcdab89*/ T_MASK ^ 0x10325476;
        c = /*0x98badcfe*/ T_MASK ^ 0x67452301;
        d = 0x10325476;
        /* Round 1. */
        /* Let [abcd k s i] denote the operation
           a = b + ((a + F(b,c,d) + X[k] + T[i]) <<< s).
        */
        /* Do the following 16 operations. */
        // Set 1
        t = a + ((b & c) | (~b & d)) + word + T1;
        a = ((t << 7) | (t >> (32 - 7))) + b;

        t = d + ((a & b) | (~a & c)) + X_1 + T2;
        d = ((t << 12) | (t >> (32 - 12))) + a;

        t = c + ((d & a) | (~d & b)) + ZERO + T3;
        c = ((t << 17) | (t >> (32 - 17))) + d;

        t = b + ((c & d) | (~c & a)) + ZERO + T4;
        b = ((t << 22) | (t >> (32 - 22))) + c;

        // Set 2
        t = a + ((b & c) | (~b & d)) + ZERO + T5;
        a = ((t << 7) | (t >> (32 - 7))) + b;

        t = d + ((a & b) | (~a & c)) + ZERO + T6;
        d = ((t << 12) | (t >> (32 - 12))) + a;

        t = c + ((d & a) | (~d & b)) + ZERO + T7;
        c = ((t << 17) | (t >> (32 - 17))) + d;

        t = b + ((c & d) | (~c & a)) + ZERO + T8;
        b = ((t << 22) | (t >> (32 - 22))) + c;

        // Set 3
        t = a + ((b & c) | (~b & d)) + ZERO + T9;
        a = ((t << 7) | (t >> (32 - 7))) + b;

        t = d + ((a & b) | (~a & c)) + ZERO + T10;
        d = ((t << 12) | (t >> (32 - 12))) + a;

        t = c + ((d & a) | (~d & b)) + ZERO + T11;
        c = ((t << 17) | (t >> (32 - 17))) + d;

        t = b + ((c & d) | (~c & a)) + ZERO + T12;
        b = ((t << 22) | (t >> (32 - 22))) + c;

        // Set 4
        t = a + ((b & c) | (~b & d)) + ZERO + T13;
        a = ((t << 7) | (t >> (32 - 7))) + b;

        t = d + ((a & b) | (~a & c)) + ZERO + T14;
        d = ((t << 12) | (t >> (32 - 12))) + a;

        t = c + ((d & a) | (~d & b)) + X_14 + T15;
        c = ((t << 17) | (t >> (32 - 17))) + d;

        t = b + ((c & d) | (~c & a)) + ZERO + T16;
        b = ((t << 22) | (t >> (32 - 22))) + c;



        /* Round 2. */
        /* Let [abcd k s i] denote the operation
           a = b + ((a + G(b,c,d) + X[k] + T[i]) <<< s).
        */
        /* Do the following 16 operations. */
        // Set 1
        t = a + ((b & d) | (c & ~d)) + X_1 + T17;
        a = ((t << 5) | (t >> (32 - 5))) + b;

        t = d + ((a & c) | (b & ~c)) + ZERO + T18;
        d = ((t << 9) | (t >> (32 - 9))) + a;

        t = c + ((d & b) | (a & ~b)) + ZERO + T19;
        c = ((t << 14) | (t >> (32 - 14))) + d;

        t = b + ((c & a) | (d & ~a)) + word + T20;
        b = ((t << 20) | (t >> (32 - 20))) + c;

        // Set 2
        t = a + ((b & d) | (c & ~d)) + ZERO + T21;
        a = ((t << 5) | (t >> (32 - 5))) + b;

        t = d + ((a & c) | (b & ~c)) + ZERO + T22;
        d = ((t << 9) | (t >> (32 - 9))) + a;

        t = c + ((d & b) | (a & ~b)) + ZERO + T23;
        c = ((t << 14) | (t >> (32 - 14))) + d;

        t = b + ((c & a) | (d & ~a)) + ZERO + T24;
        b = ((t << 20) | (t >> (32 - 20))) + c;

        // Set 3
        t = a + ((b & d) | (c & ~d)) + ZERO + T25;
        a = ((t << 5) | (t >> (32 - 5))) + b;

        t = d + ((a & c) | (b & ~c)) + X_14 + T26;
        d = ((t << 9) | (t >> (32 - 9))) + a;

        t = c + ((d & b) | (a & ~b)) + ZERO + T27;
        c = ((t << 14) | (t >> (32 - 14))) + d;

        t = b + ((c & a) | (d & ~a)) + ZERO + T28;
        b = ((t << 20) | (t >> (32 - 20))) + c;

        // Set 4
        t = a + ((b & d) | (c & ~d)) + ZERO + T29;
        a = ((t << 5) | (t >> (32 - 5))) + b;

        t = d + ((a & c) | (b & ~c)) + ZERO + T30;
        d = ((t << 9) | (t >> (32 - 9))) + a;

        t = c + ((d & b) | (a & ~b)) + ZERO + T31;
        c = ((t << 14) | (t >> (32 - 14))) + d;

        t = b + ((c & a) | (d & ~a)) + ZERO + T32;
        b = ((t << 20) | (t >> (32 - 20))) + c;




        /* Round 3. */
        /* Let [abcd k s t] denote the operation
           a = b + ((a + H(b,c,d) + X[k] + T[i]) <<< s).
        */
        /* Do the following 16 operations. */
        // Set 1
        t = a + (b ^ c ^ d) + ZERO + T33;
        a = ((t << 4) | (t >> (32 - 4))) + b;

        t = d + (a ^ b ^ c) + ZERO + T34;
        d = ((t << 11) | (t >> (32 - 11))) + a;

        t = c + (d ^ a ^ b) + ZERO + T35;
        c = ((t << 16) | (t >> (32 - 16))) + d;

        t = b + (c ^ d ^ a) + X_14 + T36;
        b = ((t << 23) | (t >> (32 - 23))) + c;

        // Set 2
        t = a + (b ^ c ^ d) + X_1 + T37;
        a = ((t << 4) | (t >> (32 - 4))) + b;

        t = d + (a ^ b ^ c) + ZERO + T38;
        d = ((t << 11) | (t >> (32 - 11))) + a;

        t = c + (d ^ a ^ b) + ZERO + T39;
        c = ((t << 16) | (t >> (32 - 16))) + d;

        t = b + (c ^ d ^ a) + ZERO + T40;
        b = ((t << 23) | (t >> (32 - 23))) + c;

        // Set 3
        t = a + (b ^ c ^ d) + ZERO + T41;
        a = ((t << 4) | (t >> (32 - 4))) + b;

        t = d + (a ^ b ^ c) + word + T42;
        d = ((t << 11) | (t >> (32 - 11))) + a;

        t = c + (d ^ a ^ b) + ZERO + T43;
        c = ((t << 16) | (t >> (32 - 16))) + d;

        t = b + (c ^ d ^ a) + ZERO + T44;
        b = ((t << 23) | (t >> (32 - 23))) + c;

        // Set 4
        t = a + (b ^ c ^ d) + ZERO + T45;
        a = ((t << 4) | (t >> (32 - 4))) + b;

        t = d + (a ^ b ^ c) + ZERO + T46;
        d = ((t << 11) | (t >> (32 - 11))) + a;

        t = c + (d ^ a ^ b) + ZERO + T47;
        c = ((t << 16) | (t >> (32 - 16))) + d;

        t = b + (c ^ d ^ a) + ZERO + T48;
        b = ((t << 23) | (t >> (32 - 23))) + c;




        /* Round 4. */
        /* Let [abcd k s t] denote the operation
           a = b + ((a + I(b,c,d) + X[k] + T[i]) <<< s).
        */
        /* Do the following 16 operations. */
        // Set 1
        t = a + (c ^ (b | ~d)) + word + T49;
        a = ((t << 6) | (t >> (32 - 6))) + b;

        t = d + (b ^ (a | ~c)) + ZERO + T50;
        d = ((t << 10) | (t >> (32 - 10))) + a;

        t = c + (a ^ (d | ~b)) + X_14 + T51;
        c = ((t << 15) | (t >> (32 - 15))) + d;

        t = b + (d ^ (c | ~a)) + ZERO + T52;
        b = ((t << 21) | (t >> (32 - 21))) + c;

        // Set 2
        t = a + (c ^ (b | ~d)) + ZERO + T53;
        a = ((t << 6) | (t >> (32 - 6))) + b;

        t = d + (b ^ (a | ~c)) + ZERO + T54;
        d = ((t << 10) | (t >> (32 - 10))) + a;

        t = c + (a ^ (d | ~b)) + ZERO + T55;
        c = ((t << 15) | (t >> (32 - 15))) + d;

        t = b + (d ^ (c | ~a)) + X_1 + T56;
        b = ((t << 21) | (t >> (32 - 21))) + c;

        // Set 3
        t = a + (c ^ (b | ~d)) + ZERO + T57;
        a = ((t << 6) | (t >> (32 - 6))) + b;

        t = d + (b ^ (a | ~c)) + ZERO + T58;
        d = ((t << 10) | (t >> (32 - 10))) + a;

        t = c + (a ^ (d | ~b)) + ZERO + T59;
        c = ((t << 15) | (t >> (32 - 15))) + d;

        t = b + (d ^ (c | ~a)) + ZERO + T60;
        b = ((t << 21) | (t >> (32 - 21))) + c;

        // Set 4
        t = a + (c ^ (b | ~d)) + ZERO + T61;
        a = ((t << 6) | (t >> (32 - 6))) + b;

        t = d + (b ^ (a | ~c)) + ZERO + T62;
        d = ((t << 10) | (t >> (32 - 10))) + a;

        t = c + (a ^ (d | ~b)) + ZERO + T63;
        c = ((t << 15) | (t >> (32 - 15))) + d;

        t = b + (d ^ (c | ~a)) + ZERO + T64;
        b = ((t << 21) | (t >> (32 - 21))) + c;


        /* Then perform the following additions. (That is increment each
           of the four registers by the value it had before this block
           was started.)
        */
        a += 0x67452301;
        b += T_MASK ^ 0x10325476;
        c += T_MASK ^ 0x67452301;
        d += 0x10325476;
        
        if (cached_hash[0]   == (md5_byte_t)(a >> 0)   &&
            cached_hash[1]   == (md5_byte_t)(a >> 8)   &&
            cached_hash[2]   == (md5_byte_t)(a >> 16)  &&
            cached_hash[3]   == (md5_byte_t)(a >> 24)  &&

            cached_hash[4]   == (md5_byte_t)(b >> 0)   &&
            cached_hash[5]   == (md5_byte_t)(b >> 8)   &&
            cached_hash[6]   == (md5_byte_t)(b >> 16)  &&
            cached_hash[7]   == (md5_byte_t)(b >> 24)  &&

            cached_hash[8]   == (md5_byte_t)(c >> 0)   &&
            cached_hash[9]   == (md5_byte_t)(c >> 8)   &&
            cached_hash[10]  == (md5_byte_t)(c >> 16)  &&
            cached_hash[11]  == (md5_byte_t)(c >> 24)  &&

            cached_hash[12]  == (md5_byte_t)(d >> 0)   &&
            cached_hash[13]  == (md5_byte_t)(d >> 8)   &&
            cached_hash[14]  == (md5_byte_t)(d >> 16)  &&
            cached_hash[15]  == (md5_byte_t)(d >> 24)
          ) {
          cuPrintf("found.\n");
          hash_word[0] = word & 0xff;
          hash_word[1] = i_1;
          hash_word[2] = i_2;
          hash_word[3] = i_3;
          // hash_word[0] = word[0];
          // hash_word[1] = word[1];
          // hash_word[2] = word[2];
          // hash_word[3] = word[3];
        }
      } // END Loop 3
    } // END Loop 2
  } // END Loop 2
}





int main (int argc, char *argv[]) {
  fprintf(stdout, "main\n");

  if (argc != 2) {
    printf("**invalid number of arguments**\n");
    return 1;
  }

  char hash_str[32];
  md5_byte_t h_hash[16];

  // Generate a MD5 hash string for the word passed in as an arg
  // The generated hash is the one we'll try to break
  create_md5_hash_str(argv[1], hash_str);


  fprintf(stdout, "original_word: |%s|\n", argv[1]);
  fprintf(stdout, "hash to break: %s\n", hash_str);

  // Split the MD5 hash into 16 8bit char chunks
  // The idea is split the 32 bit char string into 16 chunks of 2 chars each
  // Then convert the ASCII value of the pair of chars to their hexadecimal
  // The reason being because when the MD5 hash is generated instead of creting a string
  // the hash is outputed in 16 chunks of hexadecimal value
  // So to increase the speed during comparasion we convert the hash we are trying to break to 16 chunks of hexadecimal
  // values so when it comes the time to compare the generated hashes with the hash that we are tying to break
  // we only need to issue a memcmp insetad of converting the bytes to a string and doing a strcmp
  break_down_hash(h_hash, hash_str);

  hipError_t error;

  // Declare device hash
  md5_byte_t* d_hash;
  error = hipMalloc((void**)&d_hash, sizeof(md5_byte_t) * 16);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  error = hipMemcpy(d_hash, h_hash, sizeof(md5_byte_t) * 16, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  // Declare device hash
  md5_byte_t* d_word;
  error = hipMalloc((void**)&d_word, sizeof(md5_byte_t) * 4);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  cudaPrintfInit();

  do_md5<<<1,94>>>(d_hash, d_word);

  // synchronize the device and the host
  hipDeviceSynchronize();
  error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
  md5_byte_t h_word[4];
  error = hipMemcpy(h_word, d_word, sizeof(md5_byte_t) * 4, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  printf("|%c%c%c%c|\n", h_word[0], h_word[1], h_word[2], h_word[3]);

  error = hipFree(d_hash);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  error = hipFree(d_word);
  if (error != hipSuccess) {
    printf("%s:%d error: %d - %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));
  }

  printf("finished!\n");

  return 1;
}




// Create hash string for |word|
// This is the hash string to the original hash we are trying to break
// We use the hash string to create the 16 8bit hexadecinal chunks
void create_md5_hash_str(const char* word, char* hash_str) {
  int di;

  md5_state_t state;
  md5_byte_t digest[16];  

  md5_init(&state);
  md5_append(&state, (const md5_byte_t *)word, strlen(word));
  md5_finish(&state, digest);
  
  for (di = 0; di < 16; di++) {
    sprintf(hash_str + di * 2, "%02x", digest[di]); 
  }
}

void break_down_hash (md5_byte_t* hash, char* hash_str) {
  int i, j;
  int digest = 0;
  int dec1 = 0;
  int dec2 = 0;
  for (i = 0, j = 0; i < 32; i += 2, j++) {
    dec1 = hex_to_decimal(hash_str[i]);
    dec2 = hex_to_decimal(hash_str[i+1]);
    // fprintf(stdout, "dec1: %d\n", dec1);
    // fprintf(stdout, "dec2: %d\n", dec2);
    digest =  dec1 * 16 + dec2;
    hash[j] = digest;
    // fprintf(stdout, "i %d - digest: %d\n", i, digest);
  }
}

int hex_to_decimal (char c) {
  switch (c) {
    case '0':
      return 0;
    case '1':
      return 1;
    case '2':
      return 2;
    case '3':
      return 3;
    case '4':
      return 4;
    case '5':
      return 5;
    case '6':
      return 6;
    case '7':
      return 7;
    case '8':
      return 8;
    case '9':
      return 9;
    case 'a':
    case 'A':
      return 10;
    case 'b':
    case 'B':
      return 11;
    case 'c':
    case 'C':
      return 12;
    case 'd':
    case 'D':
      return 13;
    case 'e':
    case 'E':
      return 14;
    case 'f':
    case 'F':
      return 15;
    default:
      fprintf(stdout, "FAILED to get convert %c to decimal\n", c);
  }

  return -1;
}

